#include "hip/hip_runtime.h"
//FIXME: for this to work, we will need to rewrite a lot of the previous calsses to be able to be run by __devices__, and to not use certain cpp features (vecs, for example).


#include <stdio.h>
#include <stdlib.h>
#include <string.h> 
#include <vector>
#include <fstream>

#include "RAII_Classes/getInput.cpp"
#include "RAII_Classes/writeOutput.cpp"
#include "RAII_Classes/out.h"
#include "RAII_Classes/buffer.hpp"

#include "ALGS/WFAlg.h"
#include "ALGS/randomAlg.h"
#include "ALGS/optAlg.h"
#include "ALGS/greedyAlg.h"
#include "ALGS/doubleCoverageAlg.h"
#include "ALGS/KCentersAlg.h"

#define N 10000000

struct cost{
    int location;
    int rand;
    int greedy;
    int OPT;
    int WFA;
    int DC;
    int KC;
    std::vector<int> input;
};

__global__ void compute(cost *out, Mspace *metric_space, int k, int n){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid>n)
        return;
    
    RandomAlg ralg;
    GreedyAlg galg;
    OptAlg oalg;
    WFAlg walg;
    DoubleCoverageAlg dalg;
    KCentersAlg kalg;

    std::vector<int> server_locations;
    server_locations.reserve(k);
    for(int j = 0;j<k;j++)
        server_locations.push_back(j);
    
    int size = out[tid].input.size();
    ralg.setGraph(*metric_space);
    ralg.setServers(k, server_locations);
    out[tid].rand = ralg.runAlg(out[tid].input, size);

    
}